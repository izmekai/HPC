
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void CudaAddition(int* x, int* y, int* z, int N){
      int index = blockIdx.x * blockDim.x + threadIdx.x;

      if (index < N) {
          z[index] = x[index]+y[index];
      }
}

int main(){
    int N = 3;

    int *x = (int *)malloc(sizeof(int)*N);
    int *y = (int *)malloc(sizeof(int)*N);
    int *z = (int *)malloc(sizeof(int)*N);


    for(int i=0;i<N;i++){
          x[i]=1;
          y[i]=1;
    }


    int *a , *b , *c;
    hipMalloc(&a , sizeof(int)*N);
    hipMalloc(&b , sizeof(int)*N);
    hipMalloc(&c , sizeof(int)*N);

    hipMemcpy(a, x, sizeof(int)*N, hipMemcpyHostToDevice);
    hipMemcpy(b, y, sizeof(int)*N, hipMemcpyHostToDevice);

    dim3 th(N);
    CudaAddition<<<1, th>>>(a, b, c, N);

    hipMemcpy(z, c, sizeof(int)*N, hipMemcpyDeviceToHost);

    for(int i=0;i<N;i++){
        cout<<z[i]<<" ";
    }
    return 0;
}